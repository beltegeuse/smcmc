#include "hip/hip_runtime.h"
// This file is part of the reference implementation for the paper 
//   Bayesian Collaborative Denoising for Monte-Carlo Rendering
//   Malik Boughida and Tamy Boubekeur.
//   Computer Graphics Forum (Proc. EGSR 2017), vol. 36, no. 4, p. 137-153, 2017
//
// All rights reserved. Use of this source code is governed by a
// BSD-style license that can be found in the LICENSE.txt file.

#include "CudaHistogramDistance.h"

#include "CudaUtils.h"

#include <iostream>
#include <cstdlib>

using namespace std;


#define N 10

namespace bcd
{

	__global__ void testCudaPrintKernel()
	{
		printf("(%d, %d): hello\n", blockIdx.x, threadIdx.x);
	}

	void testCudaPrint()
	{
		cout << "Entering testCudaPrint()" << endl;
#if __CUDA_ARCH__ >= 200
		cout << "Cuda arch is >= 200" << endl;
#endif

		dim3 nbOfBlocksPerGrid(3);
		dim3 nbOfThreadsPerBlock(2);
		testCudaPrintKernel<<<nbOfBlocksPerGrid, nbOfThreadsPerBlock>>>();
		hipDeviceSynchronize();
	}

	/// @brief Parameters that will remain constant for all the program
	struct CudaHistogramDistanceConstantParameters
	{
	public:
		int m_histogramImageColumnOffset;
		int m_histogramImageLineOffset;
		int m_nbOfSamplesImageColumnOffset;
		int m_nbOfSamplesImageLineOffset;
		int m_patchSize;
		int m_searchWindowSize;
		int m_nbOfBins;
		int m_nbOfFloatsInHistogramPatch; // = m_patchSize * m_patchSize * m_nbOfBins
		int m_powerOfTwoBeforeNbOfFloatsInHistogramPatch; // highest power of two that is strictly inferior to m_nbOfFloatsInHistogramPatch
	};

	/* // to be tested
	/// @brief Parameters which will change for each kernel change
	struct CudaHistogramDistanceParameters
	{
	public:
		float* m_dMainPatchTopLeftCornerHist;
		float* m_dSearchWindowTopLeftCornerHist;
		float* m_dMainPatchTopLeftCornerNbOfSamples;
		float* m_dSearchWindowTopLeftCornerNbOfSamples;
	};
	*/

	__constant__ CudaHistogramDistanceConstantParameters g_constantParameters;
	//__constant__ CudaHistogramDistanceParameters g_parameters;

	__global__ void computeDistancesWithCuda(
			float* o_dOutputDistances,
			float* i_dMainPatchTopLeftCornerHist, float* i_dSearchWindowTopLeftCornerHist,
			float* i_dMainPatchTopLeftCornerNbOfSamples, float* i_dSearchWindowTopLeftCornerNbOfSamples)
	{
		// threadIdx.x: bin index
		// threadIdx.y: patch column offset
		// threadIdx.z: patch line offset
		// blockIdx.x: search window column offset
		// blockIdx.y: search window line offset

		extern __shared__ float s_dDynamicSharedMemory[];
		float* s_dSumTerms = s_dDynamicSharedMemory;
		float* s_dNonBoth0SumTerms = reinterpret_cast<float*>(s_dDynamicSharedMemory + g_constantParameters.m_nbOfFloatsInHistogramPatch);

	//	if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0)
	//		printf("(%d, %d): hello\n", blockIdx.x, blockIdx.y);

	//	if(threadIdx.x >= g_constantParameters.m_nbOfBins)
	//		return; // Warning: danger with later __syncthreads()...?!
		float h1 = i_dMainPatchTopLeftCornerHist[
				threadIdx.x +
				threadIdx.y * g_constantParameters.m_histogramImageColumnOffset +
				threadIdx.z * g_constantParameters.m_histogramImageLineOffset];
		float n1 = i_dMainPatchTopLeftCornerNbOfSamples[
				threadIdx.y * g_constantParameters.m_nbOfSamplesImageColumnOffset +
				threadIdx.z * g_constantParameters.m_nbOfSamplesImageLineOffset];
		float h2 = i_dSearchWindowTopLeftCornerHist[
				threadIdx.x +
				(blockIdx.x + threadIdx.y) * g_constantParameters.m_histogramImageColumnOffset +
				(blockIdx.y + threadIdx.z) * g_constantParameters.m_histogramImageLineOffset];
		float n2 = i_dSearchWindowTopLeftCornerNbOfSamples[
				(blockIdx.x + threadIdx.y) * g_constantParameters.m_nbOfSamplesImageColumnOffset +
				(blockIdx.y + threadIdx.z) * g_constantParameters.m_nbOfSamplesImageLineOffset];
		float diff = h1 * n2 - h2 * n1;
		float h1h2Sum = h1 + h2;
		int termIndex = threadIdx.x + g_constantParameters.m_nbOfBins *
				(threadIdx.y + g_constantParameters.m_patchSize * threadIdx.z);
		s_dNonBoth0SumTerms[termIndex] = h1h2Sum > 0 ? 1 : 0;
		s_dSumTerms[termIndex] = h1h2Sum > 0 ? (diff * diff) / (n1 * n2 * h1h2Sum) : 0;

		// now we sum up the terms
		__syncthreads();

		int nbOfTerms = g_constantParameters.m_powerOfTwoBeforeNbOfFloatsInHistogramPatch;
		if(termIndex + nbOfTerms < g_constantParameters.m_nbOfFloatsInHistogramPatch)
		{
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + nbOfTerms];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + nbOfTerms];
		}
		__syncthreads();
		nbOfTerms >>= 1;

	//	for(; nbOfTerms > 32; nbOfTerms >>= 1) // for "optimization" that does not work
		for(; nbOfTerms > 0; nbOfTerms >>= 1)
		{
			if(termIndex < nbOfTerms)
			{
				s_dSumTerms[termIndex] += s_dSumTerms[termIndex + nbOfTerms];
				s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + nbOfTerms];
				__syncthreads();
			}
		}

		/* "optimization" that... does not work
		if (termIndex < 32)
		{
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 32];
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 16];
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 8];
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 4];
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 2];
			s_dSumTerms[termIndex] += s_dSumTerms[termIndex + 1];

			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 32];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 16];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 8];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 4];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 2];
			s_dNonBoth0SumTerms[termIndex] += s_dNonBoth0SumTerms[termIndex + 1];
		}
		*/

		if(termIndex == 0)
		{
	//		printf("(%d, %d): %f\n", blockIdx.x, blockIdx.y, s_dSumTerms[0]);
			o_dOutputDistances[blockIdx.x + gridDim.x * blockIdx.y] =
					s_dSumTerms[0] / s_dNonBoth0SumTerms[0];
		}
	}


	CudaHistogramDistance::CudaHistogramDistance(
			const float* i_pHistogramData, const float* i_pNbOfSamplesData,
			int i_width, int i_height, int i_nbOfBins,
			int i_patchRadius, int i_searchWindowRadius) :
		m_dHistogramData(NULL), m_dNbOfSamplesData(NULL),
		m_dOutputDistances(NULL),
		m_width(i_width), m_height(i_height), m_nbOfBins(i_nbOfBins),
		m_patchRadius(i_patchRadius), m_searchWindowRadius(i_searchWindowRadius),
		m_patchSize(1 + 2 * i_patchRadius), m_searchWindowSize(1 + 2 * i_searchWindowRadius),
		m_nbOfFloatsInHistogramPatch(m_patchSize * m_patchSize * m_nbOfBins)
	{
		int nbOfPixels = i_width * i_height;
		int nbOfPixelsInSearchWindow = (2*i_searchWindowRadius + 1)*(2*i_searchWindowRadius + 1);
		HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&m_dOutputDistances), nbOfPixelsInSearchWindow * sizeof(float)));

		int nbOfFloatsInHistogramImage = nbOfPixels * i_nbOfBins;
		HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&m_dHistogramData), nbOfFloatsInHistogramImage * sizeof(float)));
		HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&m_dNbOfSamplesData), nbOfPixels * sizeof(float)));
		HANDLE_ERROR(hipMemcpy(m_dHistogramData, i_pHistogramData, nbOfFloatsInHistogramImage * sizeof(float), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(m_dNbOfSamplesData, i_pNbOfSamplesData, nbOfPixels * sizeof(float), hipMemcpyHostToDevice));

		CudaHistogramDistanceConstantParameters params;
		params.m_histogramImageColumnOffset = i_nbOfBins;
		params.m_histogramImageLineOffset = i_nbOfBins * i_width;
		params.m_nbOfSamplesImageColumnOffset = 1;
		params.m_nbOfSamplesImageLineOffset = i_width;
		params.m_patchSize = m_patchSize;
		params.m_searchWindowSize = m_searchWindowSize;
		params.m_nbOfBins = i_nbOfBins;
		params.m_nbOfFloatsInHistogramPatch = m_nbOfFloatsInHistogramPatch;
		params.m_powerOfTwoBeforeNbOfFloatsInHistogramPatch = previousPowerOfTwo(params.m_nbOfFloatsInHistogramPatch);
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(g_constantParameters), &params, sizeof(CudaHistogramDistanceConstantParameters)));
	}

	CudaHistogramDistance::~CudaHistogramDistance()
	{
		HANDLE_ERROR(hipFree(m_dHistogramData));
		HANDLE_ERROR(hipFree(m_dNbOfSamplesData));
		HANDLE_ERROR(hipFree(m_dOutputDistances));
	}

	int CudaHistogramDistance::previousPowerOfTwo(int i_number)
	{
		int powerOfTwo = 1;
		while(powerOfTwo < i_number)
			powerOfTwo <<= 1;
		return powerOfTwo >> 1;
	}


	void CudaHistogramDistance::computeDistances(float* o_pDistances, int i_line, int i_column)
	{
		int mainPatchTopLeftCornerOffset = m_width * (i_line - m_patchRadius) + (i_column - m_patchRadius);

		// searchWindowExtended = searchWindow + border of thickness m_patchRadius
		int searchWindowExtendedTopBorderLine = max(0, i_line - m_searchWindowRadius - m_patchRadius);
		int searchWindowExtendedLeftBorderColumn = max(0, i_column - m_searchWindowRadius - m_patchRadius);
		int searchWindowExtendedBottomBorderLine = min(m_height - 1, i_line + m_searchWindowRadius + m_patchRadius);
		int searchWindowExtendedRightBorderColumn = min(m_width - 1, i_column + m_searchWindowRadius + m_patchRadius);
		int searchWindowNbOfLines = searchWindowExtendedBottomBorderLine - searchWindowExtendedTopBorderLine - 1;
		int searchWindowNbOfColumns = searchWindowExtendedRightBorderColumn - searchWindowExtendedLeftBorderColumn - 1;
		int searchWindowExtendedTopLeftCornerOffset = m_width * searchWindowExtendedTopBorderLine + searchWindowExtendedLeftBorderColumn;
		dim3 nbOfBlocksPerGrid(searchWindowNbOfColumns, searchWindowNbOfLines);
		dim3 nbOfThreadsPerBlock(m_nbOfBins, m_patchSize, m_patchSize);

		computeDistancesWithCuda<<< nbOfBlocksPerGrid, nbOfThreadsPerBlock,
									m_nbOfFloatsInHistogramPatch * (sizeof(float) + sizeof(float)) >>>(
				m_dOutputDistances,
				m_dHistogramData + mainPatchTopLeftCornerOffset * m_nbOfBins,
				m_dHistogramData + searchWindowExtendedTopLeftCornerOffset * m_nbOfBins,
				m_dNbOfSamplesData + mainPatchTopLeftCornerOffset,
				m_dNbOfSamplesData + searchWindowExtendedTopLeftCornerOffset);

		hipDeviceSynchronize();
		HANDLE_ERROR(hipMemcpy(o_pDistances, m_dOutputDistances, m_searchWindowSize * m_searchWindowSize * sizeof(float), hipMemcpyDeviceToHost));
	}

} // namespace bcd
